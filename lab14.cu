#include <thrust/host_vector.h> 
#include <thrust/device_vector.h> 
#include <thrust/generate.h> 
#include <thrust/sort.h> 
#include <thrust/copy.h> 
#include <thrust/unique.h>
#include <thrust/extrema.h> 
#include <cstdlib> 
#include <thrust/count.h>
#include <thrust/binary_search.h>
#include <thrust/inner_product.h>
#include <iomanip>

struct rand_functor {
const int a; 
rand_functor(int _a) : a(_a) {} 
__host__ __device__ 
int  operator()() const { 
return rand() % a; 
}}; 

template <typename Vector>
void print_vector(const std::string& name, const Vector& v)
{
  typedef typename Vector::value_type T;
  std::cout << "  " << std::setw(20) << name << "  ";
  thrust::copy(v.begin(), v.end(), std::ostream_iterator<T>(std::cout, " "));
  std::cout << std::endl;
}

int main(void) { 

for (int n =2; n< (2<<20); n *=2){ 

thrust::host_vector<int> hv(n); 

thrust::generate(hv.begin(), hv.end(), rand_functor(n)); 

thrust::device_vector<int> dv = hv; 
thrust::sort(dv.begin(), dv.end()); 
thrust::counting_iterator<int> search_begin(0);

thrust::device_vector<int> B(n);
thrust::device_vector<int> C(n);

thrust::reduce_by_key(dv.begin(), dv.end(), thrust::constant_iterator<int>(1), B.begin(), C.begin());

thrust::device_vector<int>::iterator iter = 
thrust::max_element(C.begin(), C.end());

unsigned int position = iter - dv.begin(); 
float max_val = *iter;

printf("Max bin capacity for N = %d bins is: %.0f \n",n, max_val);
} 
return 0; 
}

